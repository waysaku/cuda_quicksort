#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cutil.h>

__global__ void cuda_kernel(float* device_data, float* device_out_data)
{
	int tid = threadIdx.x;
	int t = tid * 2;
	device_out_data[tid] = t;
}

int main(int argc, char** argv)
{
	CUT_DEVICE_INIT(argc, argv);

	float* host_data = (float*)malloc(sizeof(float) * 1024);
	for(int i = 0; i < 1024; i++) {
	    host_data[i] = i;
	}

	float* device_data;
	hipMalloc((void**)&device_data, sizeof(float) * 1024);
	hipMemcpy(device_data, host_data, sizeof(float) * 1024, hipMemcpyHostToDevice);

	float* device_out_data;
	hipMalloc((void**)&device_out_data, sizeof(float) * 1024);

	dim3 grid(8, 8);
	dim3 threads(4, 4);

	cuda_kernel<<< grid, threads, sizeof(float) * 1024 >>>(device_data, device_out_data);
}


