
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
__global__ void func(int *data)
{
	int i = threadIdx.x;

	data[i] = data[i] + data[i+32];

	if (i < 16) {
		data[i] = data[i] + data[i+16];
	}
	__syncthreads();
	if (i < 8) {
		data[i] = data[i] + data[i+8];
	}
	__syncthreads();
	if (i < 4) {
		data[i] = data[i] + data[i+8];
	}
	__syncthreads();
	if (i < 2) {
		data[i] = data[i] + data[i+2];
	}
	__syncthreads();
	if (i < 1) {
		data[i] = data[i] + data[i+1];
	}
}
int main()
{
	int *d_data;
	int *data = new int[512];
	hipMalloc((void**)&d_data, sizeof(int)*512);
	for (int i=0; i<512; i++) {
		data[i] = 3;
	}
	hipMemcpy(d_data, data, sizeof(int)*512, hipMemcpyHostToDevice);
	func<<<1,64>>>(d_data);
	hipMemcpy(data, d_data, sizeof(int)*512, hipMemcpyDeviceToHost);

	printf("sum = %d\n", data[0]);

	assert(data[0] == 64*3);
}


