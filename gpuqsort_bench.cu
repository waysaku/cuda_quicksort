#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cutil.h>
#include <gpuqsort.h>

void to_rondom(unsigned int *ary, unsigned int len)
{
    unsigned int i, tmp, r;
    srand( (unsigned)time(NULL));
    for( i = 0; i < len - 1; i++) {
	r = rand() % (len - i) + i;
	tmp = ary[i];
	ary[i] = ary[r];
	ary[r] = tmp;
    }
}


int main( int argc, char** argv) 
{
    unsigned int i;
    //long ary[100000000] = {};
    //unsigned int ary[10] = {};
    const unsigned int MEASURES = 5;
    //const unsigned int STARTSIZE = 2<<19;
    const unsigned int STARTSIZE = 100000000;

    unsigned int maxsize = STARTSIZE<<(MEASURES - 1);

    unsigned int* ary = new unsigned int[maxsize];
    unsigned int len = STARTSIZE;

    for( i = 0; i < len; i++) {
	ary[i] = i + 1;
    }
    to_rondom(ary, len);

    /*
    for( i = 0; i < len; i++) {
	printf("%2d\n", ary[i]);
    }
   */ 
    
    double timerValue;
    int threads   = 0;
    int maxblocks = 0;
    int sbsize    = 0;



    clock_t t = clock();
    if(gpuqsort(ary, len, &timerValue, maxblocks, threads, sbsize, 0) != 0)
    {
	printf("Error! (%s)\n",getGPUSortErrorStr());
	exit(1);
    }
    printf( "%lf\n", (double)( clock() - t ) / CLOCKS_PER_SEC ) ;







    
    /*
    for( i = 0; i < len; i++) {
	printf("%2d\n", ary[i]);
    }
    */
    return 0;
}

