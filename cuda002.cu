#include <stdio.h>
#include <cutil.h>
#include "cuda002_kernel.cu"


int main(int argc, char** argv)
{

    CUT_DEVICE_INIT(argc, argv);

    unsigned int timer = 0;
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    CUT_SAFE_CALL(cutStartTimer(timer));

    float* h_idata = (float*) malloc(sizeof(float) * 100);
    for(int i = 0; i < 100; i++)
    {
	h_idata[i] = i;
    }

    float* d_idata;
    CUDA_SAFE_CALL(hipMalloc((void**) &d_idata, sizeof(float) * 100));
    CUDA_SAFE_CALL(hipMemcpy(d_idata, h_idata, sizeof(float) * 100, hipMemcpyHostToDevice));

    float* d_odata;
    CUDA_SAFE_CALL(hipMalloc((void**) &d_odata, sizeof(float) * 100));

    dim3 grid(1, 1, 1);
    dim3 threads(100, 1, 1);

    cuda002Kernel<<< grid, threads, sizeof(float) * 100 >>>(d_idata, d_odata);

    float* h_odata = (float*) malloc(sizeof(float) * 100);
    CUDA_SAFE_CALL(hipMemcpy(h_odata, d_odata, sizeof(float) * 100, hipMemcpyDeviceToHost));

    printf("input data, output data\n");
    for(int i = 0; i < 100; i++)
    {
	printf("%f, %f\n", h_idata[i], h_odata[i]);
    }

    CUT_SAFE_CALL(cutStopTimer(timer));
    printf("Processing time: %f(ms)\n", cutGetTimerValue(timer));
    CUT_SAFE_CALL(cutDeleteTimer(timer));

    free(h_idata);
    free(h_odata);
    CUDA_SAFE_CALL(hipFree(d_idata));
    CUDA_SAFE_CALL(hipFree(d_odata));

    CUT_EXIT(argc, argv);


}


