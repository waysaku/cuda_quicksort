#include "hip/hip_runtime.h"
#include <stdio.h>
#include "sharedmem.cuh"

__global__ void cuda002Kernel(float* g_idata, float* g_odata)
{
    SharedMemory<float> smem;
    float* sdata = smem.getPointer();

    const unsigned int tid = threadIdx.x;
    sdata[tid] = g_idata[tid];
    __syncthreads();

    sdata[tid] = (float)2 * sdata[tid];
    __syncthreads();

    g_odata[tid] = sdata[tid];
}
